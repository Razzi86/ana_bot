#include <hip/hip_runtime.h>

#include <stdio.h>  // For error logging
#include <limits.h> // For INT_MAX

// TODO: Do research online and find an official standard for using CUDA

// Define constants for filter sizes and other parameters
#define FILTER_SIZE 3  // Example filter size for a smoothing operation
#define BLOCK_SIZE 16  // Define block size for kernel execution

// Struct for representing a 3D point
struct PointXYZ {
    float x, y, z;
};

// CUDA kernel to filter depth images based on a maximum depth threshold
__global__ void filterDepth(float* img, int width, int height, float maxDepth) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * width + idx;

    if (idx < width && idy < height) {
        float depth = img[index];
        if (depth > maxDepth) {
            img[index] = NAN;  // Set out of range values to NaN
        }
    }
}

// CUDA kernel to perform a simple box smoothing on the depth image
__global__ void smoothDepth(float* input, float* output, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * width + idx;

    if (idx >= width || idy >= height) return;

    float sum = 0.0f;
    int count = 0;

    // Apply a simple mean filter
    for (int ky = -FILTER_SIZE / 2; ky <= FILTER_SIZE / 2; ++ky) {
        for (int kx = -FILTER_SIZE / 2; kx <= FILTER_SIZE / 2; ++kx) {
            int n_x = idx + kx;
            int n_y = idy + ky;
            if (n_x >= 0 && n_x < width && n_y >= 0 && n_y < height) {
                sum += input[n_y * width + n_x];
                count++;
            }
        }
    }

    output[index] = sum / count;
}

// Host function to launch the filter and smoothing kernels
extern "C" void processDepthImage(float* hostImage, int width, int height, float maxDepth) {
    float* devImage;
    size_t imageSize = width * height * sizeof(float);
    float* smoothedDepth;

    // Allocate memory on the GPU
    hipMalloc((void**)&devImage, imageSize);
    hipMalloc((void**)&smoothedDepth, imageSize);

    // Copy data from host to device
    hipMemcpy(devImage, hostImage, imageSize, hipMemcpyHostToDevice);

    // Setup the execution configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Run the depth filtering kernel
    filterDepth<<<dimGrid, dimBlock>>>(devImage, width, height, maxDepth);
    // Run the smoothing kernel
    smoothDepth<<<dimGrid, dimBlock>>>(devImage, smoothedDepth, width, height);

    // Copy the processed data back to host
    hipMemcpy(hostImage, smoothedDepth, imageSize, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(devImage);
    hipFree(smoothedDepth);
}

__global__ void voxelGridDownsample(PointXYZ *input, PointXYZ *output, int numPoints, float voxelSize, int *minIndexGrid, int gridDimensions) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numPoints) return;

    int gridX = floor(input[idx].x / voxelSize);
    int gridY = floor(input[idx].y / voxelSize);
    int gridZ = floor(input[idx].z / voxelSize);
    int gridIndex = gridX + gridY * gridDimensions + gridZ * gridDimensions * gridDimensions;

    atomicMin(&minIndexGrid[gridIndex], idx);
    __syncthreads();

    if (idx == minIndexGrid[gridIndex]) {
        output[gridIndex] = input[idx];
    }
}

extern "C" void processPointCloudVoxelGrid(PointXYZ *hostPoints, int numPoints, float voxelSize) {
    PointXYZ *devPoints, *devReducedPoints;
    int *minIndexGrid;
    int gridDimensions = ceil(10.0 / voxelSize);  // Define the grid dimensions based on the voxel size
    int gridSize = gridDimensions * gridDimensions * gridDimensions;

    // Allocate memory on GPU
    hipMalloc(&devPoints, numPoints * sizeof(PointXYZ));
    hipMalloc(&devReducedPoints, gridSize * sizeof(PointXYZ));
    hipMalloc(&minIndexGrid, gridSize * sizeof(int));
    hipMemset(minIndexGrid, INT_MAX, gridSize * sizeof(int));

    // Copy data from host to device
    hipMemcpy(devPoints, hostPoints, numPoints * sizeof(PointXYZ), hipMemcpyHostToDevice);

    // Configure and launch the kernel
    dim3 block(256);
    dim3 grid((numPoints + block.x - 1) / block.x);
    voxelGridDownsample<<<grid, block>>>(devPoints, devReducedPoints, numPoints, voxelSize, minIndexGrid, gridDimensions);

    // Copy processed data back to host
    hipMemcpy(hostPoints, devReducedPoints, gridSize * sizeof(PointXYZ), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(devPoints);
    hipFree(devReducedPoints);
    hipFree(minIndexGrid);
}
