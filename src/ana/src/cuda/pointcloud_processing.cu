#include <hip/hip_runtime.h>

#include <stdio.h>  // For error logging
#include <limits.h> // For INT_MAX

// Define constants for filter sizes and other parameters
#define FILTER_SIZE 3  // Example filter size for a smoothing operation
#define BLOCK_SIZE 16  // Define block size for kernel execution

// CUDA kernel to filter depth images based on a maximum depth threshold
__global__ void filterDepth(float* img, int width, int height, float maxDepth) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * width + idx;

    if (idx < width && idy < height) {
        float depth = img[index];
        if (depth > maxDepth) {
            img[index] = NAN;  // Set out of range values to NaN
        }
    }
}

// CUDA kernel to perform a simple box smoothing on the depth image
__global__ void smoothDepth(float* input, float* output, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * width + idx;

    if (idx >= width || idy >= height) return;

    float sum = 0.0f;
    int count = 0;

    // Apply a simple mean filter
    for (int ky = -FILTER_SIZE / 2; ky <= FILTER_SIZE / 2; ++ky) {
        for (int kx = -FILTER_SIZE / 2; kx <= FILTER_SIZE / 2; ++kx) {
            int n_x = idx + kx;
            int n_y = idy + ky;
            if (n_x >= 0 && n_x < width && n_y >= 0 && n_y < height) {
                sum += input[n_y * width + n_x];
                count++;
            }
        }
    }

    output[index] = sum / count;
}

// Host function to call the kernels
extern "C" void processDepthImage(float* hostImage, int width, int height, float maxDepth) {
    float* devImage;
    size_t imageSize = width * height * sizeof(float);
    float* smoothedDepth;

    hipMalloc((void**)&devImage, imageSize);
    hipMalloc((void**)&smoothedDepth, imageSize);
    hipMemcpy(devImage, hostImage, imageSize, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Apply depth filtering
    filterDepth<<<dimGrid, dimBlock>>>(devImage, width, height, maxDepth);

    // Apply smoothing
    smoothDepth<<<dimGrid, dimBlock>>>(devImage, smoothedDepth, width, height);

    hipMemcpy(hostImage, smoothedDepth, imageSize, hipMemcpyDeviceToHost);

    hipFree(devImage);
    hipFree(smoothedDepth);
}

struct PointXYZ {
    float x, y, z;
};

__global__ void voxelGridDownsample(PointXYZ *input, PointXYZ *output, int numPoints, float voxelSize, int *minIndexGrid, int gridDimensions) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numPoints) return;

    int gridX = floor(input[idx].x / voxelSize);
    int gridY = floor(input[idx].y / voxelSize);
    int gridZ = floor(input[idx].z / voxelSize);
    int gridIndex = gridX + gridY * gridDimensions + gridZ * gridDimensions * gridDimensions;

    atomicMin(&minIndexGrid[gridIndex], idx);
    __syncthreads();

    if (idx == minIndexGrid[gridIndex]) {
        output[gridIndex] = input[idx];
    }
}

extern "C" void processPointCloudVoxelGrid(PointXYZ *hostPoints, int numPoints, float voxelSize) {
    PointXYZ *devPoints, *devReducedPoints;
    int *minIndexGrid;
    int gridDimensions = ceil(10.0 / voxelSize);  // Assuming a 10x10x10m space for example
    int gridSize = gridDimensions * gridDimensions * gridDimensions;

    hipMalloc(&devPoints, numPoints * sizeof(PointXYZ));
    hipMalloc(&devReducedPoints, gridSize * sizeof(PointXYZ));
    hipMalloc(&minIndexGrid, gridSize * sizeof(int));
    hipMemset(minIndexGrid, INT_MAX, gridSize * sizeof(int));

    hipMemcpy(devPoints, hostPoints, numPoints * sizeof(PointXYZ), hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((numPoints + block.x - 1) / block.x);
    voxelGridDownsample<<<grid, block>>>(devPoints, devReducedPoints, numPoints, voxelSize, minIndexGrid, gridDimensions);

    hipMemcpy(hostPoints, devReducedPoints, gridSize * sizeof(PointXYZ), hipMemcpyDeviceToHost);

    hipFree(devPoints);
    hipFree(devReducedPoints);
    hipFree(minIndexGrid);
}
